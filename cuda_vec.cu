#include "hip/hip_runtime.h"
#include "cuda_vec.hpp"

__global__
void kernel(double *vec, double *mtr, double *res, const int n)
{
    int tid = threadIdx.y + blockDim.y * threadIdx.y;
    double tmp = 0.0;

    if(tid < n){
        for(int i = 0; i < n; i++){
            tmp += vec[i] * mtr[i * tid + n];
        }
        res[tid] = tmp;
    }
}



void getMatMulVec(double *kM, double *kx, double *kb, double *f, double *Hf,
    int n)
{
    hipMemcpy(kx, f, sizeof(double) * n, hipMemcpyHostToDevice);
    kernel<<<n / 256 + 1, n>>>(kM, kx, kb, n);
    hipMemcpy(Hf, kb, sizeof(double) * n, hipMemcpyDeviceToHost);
}
