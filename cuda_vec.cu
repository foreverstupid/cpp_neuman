#include "hip/hip_runtime.h"
#include "cuda_vec.hpp"

__global__
void mult_kernel(double *vec, double *mtr, double *res, const int n)
{
    int tid = threadIdx.y + blockDim.y * threadIdx.y;
    double tmp = 0.0;

    if(tid < n){
        for(int i = 0; i < n; i++){
            tmp += vec[i] * mtr[i * tid + n];
        }
        res[tid] = tmp;
    }
}



void cudaMatMulVec(double *kM, double *kx, double *kb, double *f,
    double *Hf, int n)
{
    hipMemcpy(kx, f, sizeof(double) * n, hipMemcpyHostToDevice);
    mult_kernel<<<grid_size, blck_size>>>(kM, kx, kb, n);
    hipMemcpy(Hf, kb, sizeof(double) * n, hipMemcpyDeviceToHost);
}



__global__
void hank_kernel(double *vec, double *res, double step, const int n)
{
    int tid = threadIdx.y + blockDim.y * threadIdx.y;
    double tmp = 0.0;
    double x = 0.0;
    double y = step * tid;

    if(tid < n){
        for(int i = 0; i < n; i++){
            tmp += j0f(x * y) * vec[i] * x;
            x += step;
        }
        res[tid] = tmp;
    }
}



void cudaHankel(double *kx, double *kb, double *f, double *Hf,
    double step, int n)
{
    hipMemcpy(kx, f, sizeof(double) * n, hipMemcpyHostToDevice);
    hank_kernel<<<grid_size, blck_size>>>(kx, kb, step, n);
    hipMemcpy(Hf, kb, sizeof(double) * n, hipMemcpyDeviceToHost);
}
