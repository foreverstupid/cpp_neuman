#include "hip/hip_runtime.h"
#include "cuda_operations.hpp"

__global__
void hank_kernel(double *vec, double *res, double step, const int n)
{
    int tid = threadIdx.y + blockDim.y * threadIdx.y;
    double tmp = 0.0;
    double x = 0.0;
    double y = step * tid;

    if(tid < n){
        for(int i = 0; i < n; i++){
            tmp += j0f(x * y) * vec[i] * x;
            x += step;
        }
        res[tid] = tmp;
    }
}



void cudaHankel(double *kx, double *kb, double *f, double *Hf,
    double step, int n)
{
    GPU_ASSERT(hipMemcpy(kx, f, sizeof(double) * n,
        hipMemcpyHostToDevice));
    hank_kernel<<<grid_size, blck_size>>>(kx, kb, step, n);
    GPU_ASSERT(hipMemcpy(Hf, kb, sizeof(double) * n,
        hipMemcpyDeviceToHost));
}



void cudaFFTForward(hipfftHandle plan, double *f, double *cuda_f,
    hipfftDoubleComplex *Ff, int n)
{
    GPU_ASSERT(hipMemcpy(cuda_f, f, sizeof(double) * n,
        hipMemcpyHostToDevice));
    hipfftExecD2Z(plan, cuda_f, Ff);
}



void cudaFFTBackward(hipfftHandle plan, hipfftDoubleComplex *Ff,
    double *cuda_f, double *f, int n)
{
    hipfftExecZ2D(plan, Ff, cuda_f);
    GPU_ASSERT(hipMemcpy(f, cuda_f, sizeof(double) * n,
        hipMemcpyDeviceToHost));
}



__global__
void complex_mul_kernel(const hipfftDoubleComplex *f,
    const hipfftDoubleComplex *g, hipfftDoubleComplex *res, int n)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    double re = f[tid].x * g[tid].y + f[tid].y * g[tid].x;
    double im = f[tid].x * g[tid].x - f[tid].y * g[tid].y;

    res[tid].x = re;
    res[tid].y = im;
}



void cudaMultiplyComplexVecs(const hipfftDoubleComplex *f,
    const hipfftDoubleComplex *g, hipfftDoubleComplex *res, int n)
{
    complex_mul_kernel<<<grid_size, blck_size>>>(f, g, res, n);
}
